#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 Zhao Zhixu
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <stdarg.h>

#include "tl_tensor.h"
#include "tl_util.h"

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

static inline __device__ int get_index(int *ids, int ndim, int *dims)
{
     int i, id;
     for (i = 0, id = ids[0]; i < ndim-1; i++)
          id = dims[i+1] * id + ids[i+1];
     return id;
}

static inline __device__ void get_indexes(int id, int *ids, int ndim, int *dims)
{
     for (int i = ndim-1; i >=0; i--) {
          ids[i] = id % dims[i];
          id = id / dims[i];
     }
}

static inline __device__ int compute_length(int ndim, const int *dims)
{
     int i, len;

     assert(ndim > 0);
     assert(dims);
     for (i = 0, len = 1; i < ndim; i++) {
          assert(dims[i] > 0);
          len *= dims[i];
     }
     return len;
}

static inline __device__ void check_dim(int ndim, const int *dims)
{
     int i;

     assert(ndim > 0);
     assert(dims);
     for (i = 0; i < ndim; i++)
          assert(dims[i] > 0);
}

static inline void check_tensor(const tl_tensor *t)
{
     assert(t);
     assert(t->data);
     assert(is_device_mem(t->data));
     assert(t->dtype >= 0 && t->dtype < TL_DTYPE_SIZE);
     assert(t->len == compute_length(t->ndim, t->dims));
}
