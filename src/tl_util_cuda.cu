/*
 * Copyright (c) 2018 Zhao Zhixu
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <errno.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <stdarg.h>

#include "tl_util.h"

int tl_is_device_mem(const void *ptr)
{
     assert(ptr);
     hipPointerAttribute_t attributes;
     TL_CUDA_CK(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeDevice;
}


void *tl_alloc_cuda(size_t size)
{
     void *p = NULL;

     assert(size > 0);
     TL_CUDA_CK(hipMalloc(&p, size));
     assert(p);

     return p;
}

void *tl_clone_h2d(const void *src, size_t size)
{
     void *p;

     assert(src);
     p = tl_alloc_cuda(size);
     TL_CUDA_CK(hipMemcpy(p, src, size, hipMemcpyHostToDevice));
     return p;
}

void *tl_clone_d2h(const void *src, size_t size)
{
     void *p;

     assert(tl_is_device_mem(src));
     p = tl_alloc(size);
     TL_CUDA_CK(hipMemcpy(p, src, size, hipMemcpyDeviceToHost));
     return p;
}

void *tl_clone_d2d(const void *src, size_t size)
{
     void *p;

     assert(tl_is_device_mem(src));
     p = tl_alloc_cuda(size);
     TL_CUDA_CK(hipMemcpy(p, src, size, hipMemcpyDeviceToDevice));
     return p;
}

void *tl_repeat(void *data, size_t size, int times)
{
     void *p, *dst;
     int i;

     assert(data && times > 0);
     dst = p = tl_alloc(size * times);
     for (i = 0; i < times; i++, p = (char *)p + size * times)
          memmove(p, data, size);
     return dst;
}

/* The following functions are taken from APUE, the 3rd version. */
static void err_doit(int errnoflag, int error, const char *fmt, va_list ap)
{
     char buf[TL_MAXLINE];

     vsnprintf(buf, TL_MAXLINE-1, fmt, ap);
     if (errnoflag)
          snprintf(buf+strlen(buf), TL_MAXLINE-strlen(buf)-1, ": %s",
               strerror(error));
     strcat(buf, "\n");
     fflush(stdout);
     fputs(buf, stderr);
     fflush(NULL);
}

/*
 * Nonfatal error unrelated to a system call.
 * Print a message and return.
 */
void tl_err_msg(const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(0, 0, fmt, ap);
     va_end(ap);
}

/*
 * Nonfatal error unrelated to a system call.
 * Error code passed as explict parameter.
 * Print a message and return.
 */
void tl_err_cont(int error, const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(1, error, fmt, ap);
     va_end(ap);
}

/*
 * Nonfatal error related to a system call.
 * Print a message and return.
 */
void tl_err_ret(const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(1, errno, fmt, ap);
     va_end(ap);
}

/*
 * Fatal error unrelated to a system call.
 * Print a message and terminate.
 */
void tl_err_quit(const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(0, 0, fmt, ap);
     va_end(ap);
     exit(1);
}

/*
 * Fatal error unrelated to a system call.
 * Print a message, dump core, and terminate.
 */
void tl_err_bt(const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(0, 0, fmt, ap);
     va_end(ap);
     abort();
     exit(1);
}

/*
 * Fatal error unrelated to a system call.
 * Error code passed as explict parameter.
 * Print a message and terminate.
 */
void tl_err_exit(int error, const char *fmt, ...)
{
     va_list
          ap;
     va_start(ap, fmt);
     err_doit(1, error, fmt, ap);
     va_end(ap);
     exit(1);
}

/*
 * Fatal error related to a system call.
 * Print a message and terminate.
 */
void tl_err_sys(const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(1, errno, fmt, ap);
     va_end(ap);
     exit(1);
}

/*
 * Fatal error related to a system call.
 * Print a message, dump core, and terminate.
 */
void tl_err_dump(const char *fmt, ...)
{
     va_list ap;
     va_start(ap, fmt);
     err_doit(1, errno, fmt, ap);
     va_end(ap);
     abort();
/* dump core and terminate */
     exit(1);
/* shouldn’t get here */
}
